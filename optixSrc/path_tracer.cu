#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "helpers.h"
#include "path_tracer.h"
#include "random.h"
#include "BRDFUtils.h"
using namespace optix;


struct ShaderGlobals{
    float3 P;
    float3 I;
    float3 N;
    float3 Ng;
    float u, v;
    float3 dPdu;
    float3 dPdv;
};

struct PerRayData_pathtrace{
    float3 result;
    float3 radiance;
    float3 attenuation;
    float3 origin;
    float3 direction;
    float importance;
    unsigned int seed;
    int depth;
    int countEmitted;
    int done;
    int inside;
//    std::string type;
    rayType type;
};

struct PerRayData_pathtrace_shadow{
    bool inShadow;
    rayType type;
};

// Scene wide
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );

// For camera
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtBuffer<float4, 2>              output_buffer;
rtBuffer<ParallelogramLight>     lights;

rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(float3, texcoord, attribute texcoord, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );

// Environment map
rtTextureSampler<float4, 2> envmap;

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );


static __device__ inline float3 powf(float3 a, float exp){
    return make_float3(powf(a.x, exp), powf(a.y, exp), powf(a.z, exp));
}

// For miss program
rtDeclareVariable(float3,       bg_color, , );

//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

RT_PROGRAM void pathtrace_camera(){
    size_t2 screen = output_buffer.size();

    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float2 jitter_scale = inv_screen / sqrt_num_samples;
    unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
    float3 result = make_float3(0.0f);

    unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);
    do {
        unsigned int x = samples_per_pixel%sqrt_num_samples;
        unsigned int y = samples_per_pixel/sqrt_num_samples;
        float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
        float2 d = pixel + jitter*jitter_scale;
        float3 ray_origin = eye;
        float3 ray_direction = normalize(d.x*U + d.y*V + W);


        PerRayData_pathtrace prd;
        prd.result = make_float3(0.f);
        prd.attenuation = make_float3(1.f);
        prd.countEmitted = true;
        prd.done = false;
        prd.inside = false;
        prd.seed = seed;
        prd.depth = 0;
        prd.type = cameraRay;

    for(;;) {
        Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
        rtTrace(top_object, ray, prd);
        if(prd.done) {
            prd.result += prd.radiance * prd.attenuation;
            break;
        }

        // RR
        if(prd.depth >= rr_begin_depth){
            float pcont = fmaxf(prd.attenuation);
            if(rnd(prd.seed) >= pcont)
                break;
                prd.attenuation /= pcont;
            }
            prd.depth++;
            prd.result += prd.radiance * prd.attenuation;
            ray_origin = prd.origin;
            ray_direction = prd.direction;
        } // eye ray

        result += prd.result;
        seed = prd.seed;
    } while (--samples_per_pixel);

    float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);

    if (frame_number > 1){
        float a = 1.0f / (float)frame_number;
        float b = ((float)frame_number - 1.0f) * a;
        float3 old_color = make_float3(output_buffer[launch_index]);
        output_buffer[launch_index] = make_float4(a * pixel_color + b * old_color, 0.0f);
    }
    else{
        output_buffer[launch_index] = make_float4(pixel_color, 0.0f);
    }
}


// Construct the shader globals
RT_PROGRAM void constructShaderGlobals(){
    if (current_prd.depth > 10){
        current_prd.done = true;
        return;

    }
    ShaderGlobals sg;
    // Calcualte the shading and geometric normals for use with our OSL shaders
    sg.N = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    sg.Ng = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    sg.I = ray.direction;
    // The shading position
    sg.P = ray.origin + t_hit * ray.direction;
    // Texture coordinates
    sg.u = texcoord.x;
    sg.v = texcoord.y;
    float normalZ = sg.N.z;
    if (normalZ > 1.0){
        normalZ = -normalZ;
    }
    float normalX = sg.N.x;
    if (normalX < 0.0){
        normalX = -normalX;
    }
    float xzAngle = acos(dot(make_float3(sg.N.x, 0.0, normalZ), make_float3(1.0, 0.0, 0.0)));
    float xyAngle = acos(dot(make_float3(sg.N.x, sg.N.y, 0.0), make_float3(0.0, 1.0, 0.0)));
    Matrix4x4 rotx;
    rotx.setRow(0, make_float4(1, 0, 0, 0));
    rotx.setRow(1, make_float4(0, cos(xzAngle), -sin(xzAngle), 0));
    rotx.setRow(2, make_float4(0, sin(xzAngle), cos(xzAngle), 0));
    rotx.setRow(3, make_float4(0, 0, 0,1));
    
    Matrix4x4 roty;
    rotx.setRow(0, make_float4(cos(xyAngle), 0, sin(xyAngle), 0));
    rotx.setRow(1, make_float4(0, 1, 0, 0));
    rotx.setRow(2, make_float4(-sin(xyAngle), 0, cos(xyAngle), 0));
    rotx.setRow(3, make_float4(0, 0, 0,1));

//    sg.dPdu = make_float3(1.0, 0.0, 0.0) * rotx * roty;
    if (current_prd.depth > 10){
        current_prd.done = true;
        return;

    }
    current_prd.origin = ray.origin + t_hit * ray.direction;
//    metal(1, 10, optix::make_float3(1, 1, 1));
//    matte();
    ifTest();

    // Compute direct light...
    // Or shoot one...
    unsigned int num_lights = lights.size();
    float3 result = make_float3(0.0f);

    for(int i = 0; i < num_lights; ++i) {
        ParallelogramLight light = lights[i];
        float z1 = rnd(current_prd.seed);
        float z2 = rnd(current_prd.seed);
        float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

        float Ldist = length(light_pos - sg.P);
        float3 L = normalize(light_pos - sg.P);
        float nDl = dot( sg.N, L );
        float LnDl = dot( light.normal, L );
        float A = length(cross(light.v1, light.v2));

        // cast shadow ray
        if ( nDl > 0.0f && LnDl > 0.0f ) {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            shadow_prd.type = shadowRay;
            Ray shadow_ray = make_Ray( sg.P, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist );
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow){
                float weight=nDl * LnDl * A / (M_PIf*Ldist*Ldist);
                result += light.emission * weight;
            }
        }
    }
    current_prd.radiance = result;
}

__device__ void metal( float Ks,  float eta,  optix::float3 Cs ){
    ShaderGlobals sg;
    // Calcualte the shading and geometric normals for use with our OSL shaders
    sg.N = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    sg.Ng = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    sg.I = ray.direction;
    // The shading position
    sg.P = ray.origin + t_hit * ray.direction;
    // Texture coordinates
    sg.u = texcoord.x;
    sg.v = texcoord.y;

    optix::float3 $tmp1 = reflection( sg.N, eta );
    optix::float3 $tmp2 = Ks * Cs;
    current_prd.attenuation = $tmp1 * $tmp2;
}

__device__ void matte( float Kd,  float3 Cs){
    ShaderGlobals sg;
    // Calcualte the shading and geometric normals for use with our OSL shaders
    sg.N = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    sg.Ng = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    sg.I = ray.direction;
    // The shading position
    sg.P = ray.origin + t_hit * ray.direction;
    // Texture coordinates
    sg.u = texcoord.x;
    sg.v = texcoord.y;
    float3 $tmp1 = oren_nayar(sg.N, 0.5);/*ward(sg.N, normalize(cross(sg.N, make_float3(0.0, 1.0, 0.0))), 0.1, 0.1); *///phong(sg.N , 10); ///diffuse( sg.N );
    float3 $tmp2 = Kd * Cs;
    current_prd.attenuation = $tmp1 * $tmp2;
}

__device__ void ifTest( ){
    ShaderGlobals sg;
    // Calcualte the shading and geometric normals for use with our OSL shaders
    sg.N = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    sg.Ng = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    sg.I = ray.direction;
    // The shading position
    sg.P = ray.origin + t_hit * ray.direction;
    // Texture coordinates
    sg.u = texcoord.x;
    sg.v = texcoord.y;
    float ___317_e;
    float3 ___317_g;
    int ___317_q;
    float ___318_t;
    float f;
    float $const1 = 0;
    float $const2 = 1;
    char* $const3 = "diffuse";
    int $const4 = 10;
    float3 $const5 = make_float3( 5,5,5);
    int $const6 = 1;
    int $const7 = 100;
    float $const8 = 100;
    char* $const9 = "phong";
    L1:
    int $tmp1;
    if ($const1 == $const2){
        $tmp1 = 1;
    }
    else{
        $tmp1 = 0;
    }
    L2:
    if (!$tmp1){
        goto L7;
    }
    L3:
     current_prd.attenuation = diffuse( sg.N );
    L4:
    ___317_e = $const4;
    L5:
    ___317_g = $const5;
    L6:
    ___317_q = $const6;
    goto L9;
    L7:
     current_prd.attenuation = phong( sg.N,$const8 );
    L8:
    ___318_t = $const4;
    L9:
    f = $const7;
    L10:
}


rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void diffuseEmitter(){
    current_prd.radiance = current_prd.countEmitted? emission_color : make_float3(0.f);
    current_prd.done = true;
}

rtDeclareVariable(float3,        diffuse_color, , );

rtTextureSampler<float4, 2> map_texture;

RT_PROGRAM void diffuse(){
    if (current_prd.depth > 10){
        current_prd.done = true;
        return;

    }
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    float3 hitpoint = ray.origin + t_hit * ray.direction;
    current_prd.origin = hitpoint;

    float z1=rnd(current_prd.seed);
    float z2=rnd(current_prd.seed);
    float3 p;
    cosine_sample_hemisphere(z1, z2, p);
    float3 v1, v2;
    createONB(ffnormal, v1, v2);
    current_prd.direction = v1 * p.x + v2 * p.y + ffnormal * p.z;
    float3 normal_color = (normalize(world_shading_normal)*0.5f + 0.5f)*0.9;

    current_prd.attenuation = current_prd.attenuation * make_float3(tex2D(map_texture, texcoord.x, texcoord.y)); // use the diffuse_color as the diffuse response

    current_prd.countEmitted = false;

    // Compute direct light...
    // Or shoot one...
    unsigned int num_lights = lights.size();
    float3 result = make_float3(0.0f);

    for(int i = 0; i < num_lights; ++i) {
        ParallelogramLight light = lights[i];
        float z1 = rnd(current_prd.seed);
        float z2 = rnd(current_prd.seed);
        float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

        float Ldist = length(light_pos - hitpoint);
        float3 L = normalize(light_pos - hitpoint);
        float nDl = dot( ffnormal, L );
        float LnDl = dot( light.normal, L );
        float A = length(cross(light.v1, light.v2));

        // cast shadow ray
        if ( nDl > 0.0f && LnDl > 0.0f ) {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            Ray shadow_ray = make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist );
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow){
                float weight=nDl * LnDl * A / (M_PIf*Ldist*Ldist);
                result += light.emission * weight;
            }
        }
    }
    current_prd.radiance = result;
    // dctsystems.co.uk
}

rtDeclareVariable(float3,        glass_color, , );
rtDeclareVariable(float,         index_of_refraction, , );

RT_PROGRAM void glass_refract(){
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    float3 hitpoint = ray.origin + t_hit * ray.direction;
    current_prd.origin = hitpoint;
    current_prd.countEmitted = true;
    float iof;
    if (current_prd.inside) {
        // Shoot outgoing ray
        iof = 1.0f/index_of_refraction;
    }
    else {
        iof = index_of_refraction;
    }
        refract(current_prd.direction, ray.direction, ffnormal, iof);
        //prd.direction = reflect(ray.direction, ffnormal);

    if (current_prd.inside) {
        // Compute Beer's law
        current_prd.attenuation = current_prd.attenuation * powf(glass_color, 1);
    }
    current_prd.inside = !current_prd.inside;
    current_prd.radiance = make_float3(0.0f);
}

rtDeclareVariable(float, reflectivity, , );
rtDeclareVariable(int, max_depth, , );

RT_PROGRAM void reflections(){
    if (current_prd.depth > 10){
        current_prd.done = true;
        return;
    }
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
    float3 hitpoint = ray.origin + t_hit * ray.direction;

    current_prd.origin = hitpoint;

    current_prd.direction = reflect(ray.direction, ffnormal);

    unsigned int num_lights = lights.size();
    float3 result = make_float3(0.0f);

    for(int i = 0; i < num_lights; ++i) {
        ParallelogramLight light = lights[i];
        float z1 = rnd(current_prd.seed);
        float z2 = rnd(current_prd.seed);
        float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

        float Ldist = length(light_pos - hitpoint);
        float3 L = normalize(light_pos - hitpoint);
        float nDl = dot( ffnormal, L );
        float LnDl = dot( light.normal, L );
        float A = length(cross(light.v1, light.v2));

        // cast shadow ray
        if ( nDl > 0.0f && LnDl > 0.0f ) {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            Ray shadow_ray = make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist );
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow){
                float weight=nDl * LnDl * A / (M_PIf*Ldist*Ldist);
                result += light.emission * weight;
            }
        }
    }
    current_prd.radiance = result;
}
//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------
RT_PROGRAM void exception(){
    output_buffer[launch_index] = make_float4(bad_color, 0.0f);

}
//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------
RT_PROGRAM void miss(){
    current_prd.radiance = bg_color;
    current_prd.done = true;
}

RT_PROGRAM void envi_miss(){
    float theta = atan2f(ray.direction.x, ray.direction.z);
    float phi = M_PIf * 0.5f - acos(ray.direction.y);
    float u = (theta + M_PIf) * (0.5f * M_1_PIf);
    float v = 0.5f * ( 1.0f + sin(phi));
    current_prd.radiance = make_float3(tex2D(envmap, u, v));
    current_prd.done = true;
}

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow(){
    current_prd_shadow.inShadow = true;
    rtTerminateRay();
}

// OSL device function
__device__ int raytype(rayType _name){
    if (current_prd.type == _name){
        return 1;
    }
    else {
        return 0;
    }
}
//----------------------------------------------------------------------------------------------------------------------
//-------------------------------------------------------- BRDFS -------------------------------------------------------
//----------------------------------------------------------------------------------------------------------------------
__device__ optix::float3 reflection(optix::float3 _normal, float _eta){
    current_prd.direction = reflect(ray.direction, _normal);
    float cosNO = optix::dot(-_normal, eye - current_prd.origin);
    if (cosNO > 0){
        return optix::make_float3(fresnel_dielectric(cosNO, _eta));
    }
    return optix::make_float3(1.0);
}
//----------------------------------------------------------------------------------------------------------------------
__device__ optix::float3 diffuse(optix::float3 _normal){
    float z1=rnd(current_prd.seed);
    float z2=rnd(current_prd.seed);
    float3 p;
    cosine_sample_hemisphere(z1, z2, p);
    float3 v1, v2;
    createONB(_normal, v1, v2);
    current_prd.direction = v1 * p.x + v2 * p.y + _normal * p.z;
    return optix::make_float3(max(optix::dot(_normal, current_prd.direction), 0.0f) * float(M_1_PI));
}
//----------------------------------------------------------------------------------------------------------------------
__device__ optix::float3 phong(optix::float3 _normal, float _exponant){
    float z1 = rnd(current_prd.seed);
    float z2 = rnd(current_prd.seed);
    float sinTheta = sqrt(1 - pow(z1, 2 / (_exponant + 1)));
    float cosTheta= pow(z1, 1 / (_exponant + 1));
    float sinPhi = sin(2*float(M_PI)*z2);
    float cosPhi = cos(2*float(M_PI)*z2);
    current_prd.direction = make_float3 (sinTheta * cosPhi, sinTheta * sinPhi, cosTheta);

    // n + 2 / 2PI * cos^n (A);
    float3 R = reflect((current_prd.origin - eye), _normal);
    float A = optix::dot(R, current_prd.direction);

    if (A > 0){
        return make_float3((( _exponant + 2) / (2 * M_PI * pow(A, _exponant))));
    }
    return make_float3(0.0);
}
//----------------------------------------------------------------------------------------------------------------------
__device__ optix::float3 oren_nayar(float3 _normal, float _sigma){
    float z1=rnd(current_prd.seed);
    float z2=rnd(current_prd.seed);
    float3 p;
    cosine_sample_hemisphere(z1, z2, p);
    float3 v1, v2;
    createONB(_normal, v1, v2);
    float3 ray_direction = current_prd.direction;
    current_prd.direction = v1 * p.x + v2 * p.y + _normal * p.z;

    float3 L = current_prd.direction;
    float3 V = (current_prd.origin - eye);

    float A = 1 / (M_PI + ((M_PI / 2) - (2/3)) * _sigma );
    float B = _sigma / (M_PI + ( (M_PI /2) - (2/3)) * _sigma);
    float s = optix::dot(L, V) - optix::dot(_normal, L) * optix::dot(_normal, V);
    float t;
    if ( s <= 0.0){
        t = 1.0;
    }
    else{
        t = max(optix::dot(_normal, L), optix::dot(_normal, V));
    }

    return make_float3(0.8 * optix::dot(_normal, L) * (A + B * (s / t)));
}
//----------------------------------------------------------------------------------------------------------------------
__device__ optix::float3 ward(float3 _normal, float3 _t, float _xRough, float _yRough){
    float3 direction = current_prd.direction;
    float z1 = rnd(current_prd.seed);
    float z2 = rnd(current_prd.seed);
    float3 v = cross(_normal, _t);
    float3 u = cross(v, _normal);

    float phi = atan((_yRough/_xRough) * tan(2*float(M_PI)*z2));
    float theta = atan( sqrt( -log(1 - z1) / ( ((cos(phi)*cos(phi)) / (_xRough*_xRough)) + ((sin(phi)*sin(phi)) / (_yRough*_yRough)) ) ) );
    float3 h = make_float3(sin(theta)*cos(phi), sin(theta)*sin(phi), cos(theta));
    h = h.x * u + h.y * v + h.z * _normal;
    float3 o = 2 * dot(direction, h) * h + direction;
    float ps = 1.0;
    float w = ps * dot(h, direction)* pow(dot(h, _normal), 3) * sqrt(dot(o, _normal) / dot(direction, _normal));
    current_prd.direction = o;
    return make_float3((ps / ( 4 * float(M_PI) * _xRough * _yRough * sqrt(dot(current_prd.direction, _normal) * dot(direction, _normal)))) * exp(-( ((h.x / _xRough)*(h.x / _xRough)) + ((h.y / _yRough)*(h.y/_yRough)) / (dot(h,_normal)*dot(h,_normal))) ));
}

__device__ optix::float3 microfacet(char* _distribution, float3 _normal, float3 _u, float _xAlpha, float _yAlpha, float _eta, int _refract){
    if (_distribution == "beckmann" || _distribution == "Beckmann"){
        if (!_refract){
            // Generate two random numbers
            float z1 = rnd(current_prd.seed);
            float z2 = rnd(current_prd.seed);

            // Generate our microsurface normal m
            float3 m;
            float a2 = _xAlpha * _xAlpha;
            float theta = atan(sqrt(-a2 * log(1 - z1)));
            float phi = 2 * M_PI * z2;
            m = optix::normalize(make_float3(sin(theta)*cos(phi), sin(theta)*sin(phi), cos(theta)));
//            printf("m.x: %f, m.y: %f, m.z: %f\n", m.x, m.y, m.z);

            float3 direction = current_prd.direction;

            // Set the new ray direction
            current_prd.direction = 2 * optix::dot(m, current_prd.origin - eye) * m - (current_prd.origin - eye);
            float chi;
            if (optix::dot(m, _normal) <= 0.0){
                chi = 0.0;
            }
            else{
                chi = 1.0;
            }

            // Calculate the fresnel term
            float F = fresnel_dielectric(optix::dot(m, (current_prd.origin - eye)), _eta);

            // Calculate the distribution
            float D;
            float mLen, nLen;
            mLen = sqrt((m.x*m.x) + (m.y*m.y) + (m.z*m.z));
            nLen = sqrt((_normal.x*_normal.x) + (_normal.y*_normal.y) + (_normal.z*_normal.z));
            float thetaM = optix::dot(m, _normal) / (mLen*nLen);
            D  = chi / (float(M_PI) * -a2 * pow(cos(thetaM), 4));
            D *= exp(-(tan(thetaM)*tan(thetaM)) / a2);
//            printf("%f\n", D);

            // Calculate the geometric distribution
            float G1 = beckmannDistibution(direction, m, _normal, _xAlpha);
            float G2 = beckmannDistibution(current_prd.direction, m, _normal, _xAlpha);
            float G = G1 * G2;

            return make_float3((F * G * D) / (4 * optix::dot(current_prd.origin - eye, _normal) * optix::dot(current_prd.direction, _normal)));
//            return make_float3(D);
        }
    }
}

__device__ float beckmannDistibution(float3 _v, float3 _m, float3 _normal, float _a){
    float chi;
    if (optix::dot(_v, _m) / optix::dot(_v, _normal) <= 0.0){
        chi = 0.0;
    }
    else{
        chi = 1.0;
    }
    float vLen = sqrt((_v.x*_v.x) + (_v.y*_v.y) + (_v.z*_v.z));
    float nLen = sqrt((_normal.x*_normal.x) + (_normal.y*_normal.y) + (_normal.z*_normal.z));
    float thetaV = optix::dot(_v, _normal) / (vLen * nLen);
    float a = 1.0 / ( _a * tan(thetaV));//(optix::dot(_normal, (current_prd.origin - eye)) /( _a * sqrt(1 - (optix::dot(_normal, (current_prd.origin - eye))* optix::dot(_normal, (current_prd.origin - eye))))));
    float G1;
    // Supposedly cheaper
    if (a < 1.6){
        G1 = chi * ((3.535*a + 2.18*a*a) / (1 + 2.276*a + 2.577*a*a));
    }
    else{
        G1 = chi;
    }

    return G1;
}
