#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "helpers.h"
#include "Core/path_tracer.h"
#include "Core/random.h"
#include "BRDFUtils.h"

// Scene wide
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );

// For camera
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtBuffer<float4, 2>              output_buffer;
rtBuffer<ParallelogramLight>     lights;

rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );
rtDeclareVariable(unsigned int,  maxDepth      , , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, tangent, attribute tangent, );
rtDeclareVariable(float3, bitangent, attribute bitangent, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );

// Environment map
rtTextureSampler<float4, 2> envmap;


rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );


//static __device__ inline float3 powf(float3 a, float exp){
//    return make_float3(powf(a.x, exp), powf(a.y, exp), powf(a.z, exp));
//}

// For miss program
rtDeclareVariable(float3,       bg_color, , );

//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------
RT_PROGRAM void pathtrace_camera(){
    size_t2 screen = output_buffer.size();

    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float2 jitter_scale = inv_screen / sqrt_num_samples;
    unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
    float3 result = make_float3(0.0f);

    unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);
    do {
        unsigned int x = samples_per_pixel%sqrt_num_samples;
        unsigned int y = samples_per_pixel/sqrt_num_samples;
        float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
        float2 d = pixel + jitter*jitter_scale;
        float3 ray_origin = eye;
        float3 ray_direction = normalize(d.x*U + d.y*V + W);


        PerRayData_pathtrace prd;
        prd.result = make_float3(0.f);
        prd.attenuation = make_float3(1.0);
        prd.radiance = make_float3(0.0);
        prd.countEmitted = true;
        prd.done = false;
        prd.seed = seed;
        prd.depth = 0;

        Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
        rtTrace(top_object, ray, prd);

        result += prd.result;
        seed = prd.seed;
    } while (--samples_per_pixel);

    float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);

    if (frame_number > 1){
        float a = 1.0f / (float)frame_number;
        float b = ((float)frame_number - 1.0f) * a;
        float3 old_color = make_float3(output_buffer[launch_index]);
        output_buffer[launch_index] = make_float4(a * pixel_color + b * old_color, 0.0f);
    }
    else{
        output_buffer[launch_index] = make_float4(pixel_color, 0.0f);
    }
}
rtDeclareVariable(float, aperture_radius, , );
rtDeclareVariable(float3, focal_point, , );
RT_PROGRAM void depth_of_field_camera(){

    size_t2 screen = output_buffer.size();

    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float2 jitter_scale = inv_screen / sqrt_num_samples;
    unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
    float3 result = make_float3(0.0f);

    unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);
    do {
        unsigned int x = samples_per_pixel%sqrt_num_samples;
        unsigned int y = samples_per_pixel/sqrt_num_samples;
        float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
        float2 d = pixel + jitter*jitter_scale;

        float3 ray_origin = eye;
        float3 ray_direction = d.x*U + d.y*V + W;
        float focal_scale = length(focal_point-eye);


        float3 p = eye + focal_scale * normalize(d.x*U + d.y*V + W);
        float2 sample = optix::square_to_disk(make_float2(jitter.x, jitter.y));

        ray_origin = ray_origin + aperture_radius * ( sample.x * normalize( U ) +  sample.y * normalize( V ) );
        ray_direction = normalize(p - ray_origin);

        PerRayData_pathtrace prd;
        prd.result = make_float3(0.f);
        prd.attenuation = make_float3(1.0);
        prd.radiance = make_float3(0.0);
        prd.countEmitted = true;
        prd.done = false;
        prd.seed = seed;
        prd.depth = 0;

        Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
        rtTrace(top_object, ray, prd);

        result += prd.result;
        seed = prd.seed;
    } while (--samples_per_pixel);

    float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);

    if (frame_number > 1){
        float a = 1.0f / (float)frame_number;
        float b = ((float)frame_number - 1.0f) * a;
        float3 old_color = make_float3(output_buffer[launch_index]);
        output_buffer[launch_index] = make_float4(a * pixel_color + b * old_color, 0.0f);
    }
    else{
        output_buffer[launch_index] = make_float4(pixel_color, 0.0f);
    }
}

//-----------------------------------------------------------------------------
RT_PROGRAM void defaultMaterial(){
    if (current_prd.depth > 5){
        current_prd.done = true;
        return;

    }    
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    float3 hitpoint = ray.origin + t_hit * ray.direction;

    float z1=rnd(current_prd.seed);
    float z2=rnd(current_prd.seed);
    float3 p;
    cosine_sample_hemisphere(z1, z2, p);
    float3 v1, v2;
    createONB(ffnormal, v1, v2);
    float3 ray_origin = hitpoint;
    float3 ray_direction = v1 * p.x + v2 * p.y + ffnormal * p.z;

    // Compute attenuation
    current_prd.attenuation = current_prd.attenuation;
    // Compute radiance
    // Compute direct lighting for environment map

    float3 result = make_float3(0.0f);

//    float3 randDirection;
//    z1=rnd(current_prd.seed);
//    z2=rnd(current_prd.seed);
//    cosine_sample_hemisphere(z1, z2, p);
//    createONB(ffnormal, v1, v2);
//    randDirection = v1 * p.x + v2 * p.y + ffnormal * p.z;

//    float theta = atan2f(randDirection.x, randDirection.z);
//    float phi = M_PIf * 0.5f - acos(ray.direction.y);
//    float u = (theta + M_PIf) * (0.5f * M_1_PIf);
//    float v = 0.5f * ( 1.0f + sin(phi));

//    PerRayData_pathtrace_shadow shadow_prd;
//    shadow_prd.inShadow = false;
//    Ray shadow_ray = make_Ray( hitpoint, randDirection, pathtrace_shadow_ray_type, scene_epsilon, RT_DEFAULT_MAX );
//    rtTrace(top_object, shadow_ray, shadow_prd);

//    if(!shadow_prd.inShadow){
//      result += make_float3(tex2D(envmap, u, v));
//    }

    // Now for the lights in the scene
    unsigned int num_lights = lights.size();


    for(int i = 0; i < num_lights; ++i)
    {
      ParallelogramLight light = lights[i];
      float z1 = rnd(current_prd.seed);
      float z2 = rnd(current_prd.seed);
      float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

      float Ldist = length(light_pos - hitpoint);
      float3 L = normalize(light_pos - hitpoint);
      float nDl = dot( ffnormal, L );
      float LnDl = dot( light.normal, L );
      float LnDlinverse = dot(-light.normal, L);
      float A = length(cross(light.v1, light.v2));

      // cast shadow ray
      if ( nDl > 0.0f && LnDl > 0.0f )
      {
        PerRayData_pathtrace_shadow shadow_prd;
        shadow_prd.inShadow = false;
        Ray shadow_ray = make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist );
        rtTrace(top_object, shadow_ray, shadow_prd);

        if(!shadow_prd.inShadow)
        {
          float weight=nDl * LnDl * A / (M_PIf*Ldist*Ldist);
          result += light.emission * weight;
        }
      }

      // cast shadow ray for other side of light
      if ( nDl > 0.0f && LnDlinverse > 0.0f )
      {
        PerRayData_pathtrace_shadow shadow_prd;
        shadow_prd.inShadow = false;
        Ray shadow_ray = make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist );
        rtTrace(top_object, shadow_ray, shadow_prd);

        if(!shadow_prd.inShadow)
        {
          float weight=nDl * LnDlinverse * A / (M_PIf*Ldist*Ldist);
          result += light.emission * weight;
        }
      }
    }

    current_prd.radiance = result;
    current_prd.countEmitted = false;

    PerRayData_pathtrace prd;
    prd.result = current_prd.result + (current_prd.attenuation * current_prd.radiance);
    prd.attenuation = current_prd.attenuation;
    prd.seed = current_prd.seed;
    prd.depth = current_prd.depth+1;


    Ray default_ray = make_Ray( ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX );
    rtTrace(top_object, default_ray, prd);

    current_prd.result = prd.result;
    current_prd.done = true; // end the ray comming in
}
//-----------------------------------------------------------------------------
rtDeclareVariable(float3, emission_color, , );
//-----------------------------------------------------------------------------
RT_PROGRAM void diffuseEmitter(){
    if(current_prd.countEmitted){
        current_prd.result = emission_color;
    }
    current_prd.done = true;
}
//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------
RT_PROGRAM void exception(){
    rtPrintExceptionDetails();
    output_buffer[launch_index] = make_float4(bad_color, 0.0f);
}
//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------
RT_PROGRAM void miss(){
    current_prd.result = make_float3(0.0, 0.0, 0.0);
    current_prd.done = true;
}
//-----------------------------------------------------------------------------
rtDeclareVariable(float, strength, , );
rtDeclareVariable(Matrix4x4, cameraMatrix, , );
//-----------------------------------------------------------------------------
RT_PROGRAM void envi_miss(){
    float3 ray_direction = make_float3(cameraMatrix * make_float4(ray.direction.x, ray.direction.y, ray.direction.z, 1.0));
    float theta = atan2f(ray_direction.x, ray_direction.z);
    float phi = M_PIf * 0.5f - acos(ray_direction.y);
    float u = (theta + M_PIf) * (0.5f * M_1_PIf);
    float v = 0.5f * ( 1.0f + sin(phi));
    if(current_prd.countEmitted){
        current_prd.result = strength * make_float3(tex2D(envmap, u, v));
    }
    else{
        current_prd.result +=  strength * make_float3(tex2D(envmap, u, v));
    }
    current_prd.done = true;
}
//-----------------------------------------------------------------------------
rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );
//-----------------------------------------------------------------------------
RT_PROGRAM void shadow(){
    current_prd_shadow.inShadow = true;
    rtTerminateRay();
}
//-----------------------------------------------------------------------------
