#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float4, sphere, , );

rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(int, lgt_idx, attribute lgt_idx, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect_sphere(int primIdx){
    float3 center = make_float3( sphere.x, sphere.y, sphere.z );
    float radius = sphere.w;
    float3 O = ray.origin - center;
    float a = dot( ray.direction, ray.direction);
    float b = 2.0 * dot( ray.direction, O );
    float c = dot(O , O) - (radius*radius);
    float disc = b*b - (4 * a * c);
    if( disc > 0.0f ) {
        float sdisc = sqrtf( disc );
        float root1 = (-b - sdisc) / (2 * a);
        bool check_second = true;
        if( rtPotentialIntersection( root1 ) ) {
            shading_normal = geometric_normal = ((O + root1*ray.direction))/radius;
            if( rtReportIntersection( 0 ) ) check_second = false;
        }
        if( check_second ) {
            float root2 = (-b + sdisc) / ( 2 * a);
            if( rtPotentialIntersection( root2 ) ) {
                shading_normal = geometric_normal =((O + root2*ray.direction))/radius;
                rtReportIntersection( 0 );
            }
        }
    }
}

RT_PROGRAM void bounds_sphere(int, float result[6])
{
    float3 center = make_float3(sphere.x,sphere.y,sphere.z);
    float3 radiusV3 = make_float3(sphere.w,sphere.w,sphere.w);
    float3 min = center - radiusV3;
    float3 max = center + radiusV3;

    result[0] = min.x;
    result[1] = min.y;
    result[2] = min.z;
    result[3] = max.x;
    result[4] = max.y;
    result[5] = max.z;
}

