#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

// This is to be plugged into an RTgeometry object to represent
// a triangle mesh with a vertex buffer of triangle soup (triangle list)
// with an interleaved position, normal, texturecoordinate layout.

rtBuffer<float3> vertex_buffer;     
rtBuffer<float3> normal_buffer;
rtBuffer<float2> texcoord_buffer;
rtBuffer<float3> tangent_buffer;
rtBuffer<float3> bitangent_buffer;


rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, tangent, attribute tangent, );
rtDeclareVariable(float3, bitangent, attribute bitangent, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void mesh_intersect( int primIdx )
{
  int3 v_idx;
  v_idx.x = primIdx*3;
  v_idx.y = v_idx.x+1;
  v_idx.z = v_idx.y+1;

  float3 p0 = vertex_buffer[ v_idx.x];
  float3 p1 = vertex_buffer[ v_idx.y ];
  float3 p2 = vertex_buffer[ v_idx.z ];

  // Intersect ray with triangle
  float3 n;
  float  t, beta, gamma;
  if( intersect_triangle( ray, p0, p1, p2, n, t, beta, gamma ) ) {

    if(  rtPotentialIntersection( t ) ) {

      if ( normal_buffer.size() == 0 || v_idx.x < 0 || v_idx.y < 0 || v_idx.z < 0 ) {
        shading_normal = normalize( n );
      } else {
        float3 n0 = normal_buffer[ v_idx.x ];
        float3 n1 = normal_buffer[ v_idx.y ];
        float3 n2 = normal_buffer[ v_idx.z ];
        shading_normal = normalize( n1*beta + n2*gamma + n0*(1.0f-beta-gamma) );
      }
      geometric_normal = normalize( n );

      if ( texcoord_buffer.size() == 0 || v_idx.x < 0 || v_idx.y < 0 || v_idx.z < 0 ) {
        texcoord = make_float3( 1.0f, 0.0f, 0.0f );
      } else {
        float2 t0 = texcoord_buffer[ v_idx.x ];
        float2 t1 = texcoord_buffer[ v_idx.y ];
        float2 t2 = texcoord_buffer[ v_idx.z ];
        texcoord = make_float3( t1*beta + t2*gamma + t0*(1.0f-beta-gamma) );
      }

      if( tangent_buffer.size() == 0 || v_idx.x < 0 || v_idx.y < 0 || v_idx.z < 0) {
          tangent = make_float3(0.0f, 0.0f, 0.0f);
      }
      else{
          float3 tan0 = tangent_buffer[ v_idx.x ];
          float3 tan1 = tangent_buffer[ v_idx.y ];
          float3 tan2 = tangent_buffer[ v_idx.z ];
          tangent = normalize( tan1*beta + tan2*gamma + tan0*(1.0f-beta-gamma) );
      }

      if( bitangent_buffer.size() == 0 || v_idx.x < 0 || v_idx.y < 0 || v_idx.z < 0) {
          bitangent = make_float3(0.0f, 0.0f, 0.0f);
      }
      else{
          float3 bitan0 = bitangent_buffer[ v_idx.x ];
          float3 bitan1 = bitangent_buffer[ v_idx.y ];
          float3 bitan2 = bitangent_buffer[ v_idx.z ];
          bitangent = normalize( bitan1*beta + bitan2*gamma + bitan0*(1.0f-beta-gamma) );
      }

      rtReportIntersection(0u);
    }
  }
}

RT_PROGRAM void mesh_bounds (int primIdx, float result[6])
{
  int3 v_idx;
  v_idx.x = primIdx*3;
  v_idx.y = v_idx.x+1;
  v_idx.z = v_idx.y+1;

  const float3 v0   = vertex_buffer[ v_idx.x ];
  const float3 v1   = vertex_buffer[ v_idx.y ];
  const float3 v2   = vertex_buffer[ v_idx.z ];
  const float  area = length(cross(v1-v0, v2-v0));

  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if(area > 0.0f && !isinf(area)) {
    aabb->m_min = fminf( fminf( v0, v1), v2 );
    aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
  } else {
    aabb->invalidate();
  }
}

